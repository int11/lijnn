# define DELLEXPORT extern "C"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#define THREADS_PER_BLOCK 1024
// nvcc -Xcompiler -fPIC -shared -g cuda_test.cu -o cuda_test.so
using namespace std ;
const double e = 2.7182818284590452353602874713527;



__global__ void tanh_impl_Kernel(double * d_in, double * d_out, int size){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if(idx < size){
		// printf("%f\n", d_in[idx]);
		double sinh = (1 - pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]));
		double cosh = (1 + pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]));
		d_out[idx] = sinh / cosh;
		// d_out[idx] = ((1 - pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]))) / ((1 + pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx])));
	}
}

DELLEXPORT void ROIPooling2D(bool cuda, double * h_in, double * h_out, int arr_size){
	const long long int ARRAY_BYTES = arr_size * sizeof(double);
	double *d_in, *d_out;

	hipMalloc(&d_in, ARRAY_BYTES);
	hipMalloc(&d_out, ARRAY_BYTES);

 	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	int block = ceil((float)arr_size/THREADS_PER_BLOCK);
	tanh_impl_Kernel<<< block, THREADS_PER_BLOCK >>>(d_in, d_out, arr_size);
	hipDeviceSynchronize();
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}

DELLEXPORT void Cfree(void * a){
	free(a);
}