#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "cuda.cuh"
using namespace std ;


void tanh_impl(double * h_in, double * h_out, int arr_size){
	const long long int ARRAY_BYTES = arr_size * sizeof(double);
	double *d_in, *d_out;

	hipMalloc(&d_in, ARRAY_BYTES);
	hipMalloc(&d_out, ARRAY_BYTES);

 	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	int block = ceil((float)arr_size/THREADS_PER_BLOCK);
	tanh_impl_Kernel<<< block, THREADS_PER_BLOCK >>>(d_in, d_out, arr_size);
	hipDeviceSynchronize();
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}