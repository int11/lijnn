#include <iostream>
#include <hip/hip_runtime.h>

using namespace std ;

# define DELLEXPORT extern "C"


#include <iostream>

#include <cmath>
#define THREADS_PER_BLOCK 512
// nvcc -Xcompiler -fPIC -shared -g cuda_test.cu -o cuda_test.so
using namespace std ;
const double e = 2.7182818284590452353602874713527;



__global__ void tanh_impl_Kernel(double * d_in, double * d_out){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
	// double sinh = (1 - pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]));
    // double cosh = (1 + pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]));
    // d_out[idx] = sinh / cosh;
    d_out[idx] = ((1 - pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx]))) / ((1 + pow(e, (-2 * d_in[idx]))) / (2 * pow(e, -d_in[idx])));

}


DELLEXPORT double* tanh_impl(double * h_in, int arr_size){
	
	const long long int ARRAY_BYTES = arr_size * sizeof(double);
    double * h_out = (double *)malloc(arr_size);
	double *d_in, *d_out;

	hipMalloc(&d_in, ARRAY_BYTES);
	hipMalloc(&d_out, ARRAY_BYTES);

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	
	tanh_impl_Kernel<<< 1, arr_size >>>(d_in, d_out);

	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
    return h_out;
}


// int main()
// {
	
	
// 	int cnt = 100000;

// 	double * x = new double[cnt];

// 	for(int i=0; i<cnt; ++i){
// 		x[i] = 1.;
// 	}
// 	printf("a %f CPU.\n", x[0]);
// 	double* a = tanh_impl(x, cnt);

// 	printf("a %f CPU.\n", a[1]);

// 	return 0;
// }